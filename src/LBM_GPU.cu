#include "hip/hip_runtime.h"
#include "LBM_GPU.cuh"
ifstream fin_GPU("in_GPU.txt");

ofstream fout_GPU("out_GPU.dat");
ofstream fout_GPU_Cd("out_GPU_Cd.dat");
ofstream fout_GPU_Ux0("out_GPU_Ux0.dat");
ofstream fout_GPU_Ux("out_GPU_Ux.dat");

LBM_GPU::LBM_GPU()
{
// ============================================================================ //
//  LOAD THE PARAMETERS
// ============================================================================ //
	fin_GPU >> nx;				fin_GPU >> comment;
	fin_GPU >> ny;				fin_GPU >> comment;
	fin_GPU >> Lx;				fin_GPU >> comment;
	fin_GPU >> Ly;				fin_GPU >> comment;
	fin_GPU >> a;				fin_GPU >> comment;
	fin_GPU >> rho1;			fin_GPU >> comment;
	fin_GPU >> BLOCK_SIZE_X;	fin_GPU >> comment;
	fin_GPU >> BLOCK_SIZE_Y;	fin_GPU >> comment;
	fin_GPU >> BLOCK_SIZE_Z;	fin_GPU >> comment;
	fin_GPU >> D;				fin_GPU >> comment;
	fin_GPU >> Um_p;			fin_GPU >> comment;
	fin_GPU >> tau;				fin_GPU >> comment;
	fin_GPU >> nu_p;			fin_GPU >> comment;
// ============================================================================ //

	
// ============================================================================ //
//  NEW & CUDAMALLOC
// ============================================================================ //
	is_boundary_node = new int[nx*ny];	hipMalloc((void**)&d_is_boundary_node, nx*ny * sizeof(int));
	is_solid_node = new int[nx*ny];		hipMalloc((void**)&d_is_solid_node, nx*ny * sizeof(int));
	is_solid_near_node = new int[nx*ny];

	U = new float[nx*ny];				hipMalloc((void**)&d_U, nx*ny * sizeof(float));
	Ux = new float[nx*ny];				hipMalloc((void**)&d_Ux, nx*ny * sizeof(float));
	Uy = new float[nx*ny];				hipMalloc((void**)&d_Uy, nx*ny * sizeof(float));
	rho = new float[nx*ny];				hipMalloc((void**)&d_rho, nx*ny * sizeof(float));

	UN = new float[nx*ny];				hipMalloc((void**)&d_UN, nx*ny * sizeof(float));
	UxN = new float[nx*ny];				hipMalloc((void**)&d_UxN, nx*ny * sizeof(float));
	UyN = new float[nx*ny];				hipMalloc((void**)&d_UyN, nx*ny * sizeof(float));
	rhoN = new float[nx*ny];			hipMalloc((void**)&d_rhoN, nx*ny * sizeof(float));
	f = new float[nx*ny*a];				hipMalloc((void**)&d_f, nx*ny*a * sizeof(float));
	ftemp = new float[nx*ny*a];			hipMalloc((void**)&d_ftemp, nx*ny*a * sizeof(float));
	fN = new float[nx*ny*a];			hipMalloc((void**)&d_fN, nx*ny*a * sizeof(float));
	feq = new float[nx*ny*a];			hipMalloc((void**)&d_feq, nx*ny*a * sizeof(float));
	ex = new float[a];					hipMalloc((void**)&d_ex, a * sizeof(float));
	ey = new float[a];					hipMalloc((void**)&d_ey, a * sizeof(float));
	U_p = new float[nx*ny];
	Ux_p = new float[nx*ny];
	Uy_p = new float[nx*ny];
	P = new float[nx*ny];
	
	Ux0_p = new float[ny];
	Ux0 = new float[ny];				hipMalloc((void**)&d_Ux0, ny * sizeof(float));
// ============================================================================ //


// ============================================================================ //
//  MICROSCOPIC VELOCITY
// ============================================================================ //
	ex[0] = 0.0,	ey[0] = 0.0;
	ex[1] = 1.0,	ey[1] = 0.0;
	ex[2] = 0.0,	ey[2] = 1.0;
	ex[3] = -1.0,	ey[3] = 0.0;
	ex[4] = 0.0,	ey[4] = -1.0;
	ex[5] = 1.0,	ey[5] = 1.0;
	ex[6] = -1.0,	ey[6] = 1.0;
	ex[7] = -1.0,	ey[7] = -1.0;
	ex[8] = 1.0,	ey[8] = -1.0;
	hipMemcpy(d_ex, ex, a * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_ey, ey, a * sizeof(float), hipMemcpyHostToDevice);
// ============================================================================ //



// ============================================================================ //
//  SET BOUNDARY NODE
// ============================================================================ //
	sIm = nx / Lx * 0.15;
	sIM = nx / Lx * (0.15 + D) - 1;
	sJm = ny / Ly * 0.15;
	sJM = ny / Ly * (0.15 + D) - 1;

	snx = (sIM - sIm) + 1;
	sny = (sJM - sJm) + 1; 

	sn = 0;
	
	ic = (float)sIm + ((float)sIM - (float)sIm) / 2;
	jc = (float)sJm + ((float)sJM - (float)sJm) / 2;

	r = ((float)sIM - (float)sIm) / 2;

	cout << "sIm = " << sIm << endl;
	cout << "sIM = " << sIM << endl;
	cout << "sJm = " << sJm << endl;
	cout << "sJM = " << sJM << endl;
	cout << "snx = " << snx << endl;
	cout << "sny = " << sny << endl;
	cout << "ic = " << ic << endl;
	cout << "jc = " << jc << endl;
	cout << "r = " << r << endl;

	//set boundary node
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {
			if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1) is_boundary_node[i + nx*j] = 1;
			else is_boundary_node[i + nx*j] = 0;

		}
	}


	//Binary data
	/*for (i = 0; i < nx; i++) {
		for (j = ny - 1; j > -1; j--) {

			if ((i >= sIm && i <= sIM) && (j >= sJm && j <= sJM)) fin_grid_GPU >> is_solid_node[i + nx*j];
			else is_solid_node[i + nx*j] = 0;

			if (is_solid_node[i + nx*j]) sn = sn + 1;
		}
	}*/

	//set solid node
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {

			dist = sqrt(pow((float)i - ic, 2) + pow((float)j - jc, 2));
			if (dist <= r) is_solid_node[i + nx*j] = 1;
			else is_solid_node[i + nx*j] = 0;

			if (is_solid_node[i + nx*j]) sn = sn + 1;

		}
	}

	//set near solid node
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {

			is_solid_near_node[i + nx*j] = 0;

			in = i - 1;
			ip = i + 1;
			jn = j - 1;
			jp = j + 1;

			if (!is_boundary_node[i + nx*j]) {
				if (!is_solid_node[i + nx*j]) {


					if (is_solid_node[ip + nx*j]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[i + nx*jp]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[in + nx*j]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[i + nx*jn]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[ip + nx*jp]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[in + nx*jp]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[in + nx*jn]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					else if (is_solid_node[ip + nx*jn]) {
						is_solid_near_node[i + nx*j] = 1;
					}
					

				}
			}
		}
	}


	hipMemcpy(d_is_boundary_node, is_boundary_node, nx*ny * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_is_solid_node, is_solid_node, nx*ny * sizeof(int), hipMemcpyHostToDevice);
// ============================================================================ //




// ============================================================================ //
//  SET PARAMETERS & INITIAL CONDITION
// ============================================================================ //
	del_x = 1.0;
	del_y = 1.0;
	del_t = 1.0;

	c = del_y / del_t;
	c_s = (1.0 / sqrt(3.0))*c;
	del_x_p = D / (float)snx;
	del_y_p = D / (float)sny;

//	del_t_p = pow(del_y_p, 2);
//	del_t_p = 0.000013;



	//Uniform
	/*nu_p = 0.06 * (del_y_p / del_t_p) * D / Re;
	nu = (del_t_p / pow(del_y_p, 2))*nu_p;
	tau = (1.0 / pow(c_s, 2))*nu + (0.5*del_t);*/

	//Input Reynolds number and del_t
	//Um = Um_p * (del_t_p / del_y_p);
	//nu_p = (2.0 / 3.0) * Um_p * D / Re;
	//nu = (del_t_p / pow(del_y_p, 2))*nu_p;
	//tau = (1.0 / pow(c_s, 2))*nu + (0.5*del_t);

	
	//Input tau and kinematic viscosity
	del_t_p = pow(c_s, 2)*(tau - 0.5)*pow(del_y_p, 2) / nu_p;
	Re = (2.0 / 3.0) * Um_p * D / nu_p;
	Um = Um_p * del_t_p / del_y_p;
	nu = nu_p * del_t_p / pow(del_y_p, 2);




	cout << endl;
	cout << "// =================== Stability condition ================ //" << endl;
	cout << "Check 1. [tau > 0.5]" << endl;
	cout << "tau = " << tau << endl;
	cout << "Check 2. Mach number condition [Ma = Uavg/c_s << 1]" << endl;
	cout << "Ma = " << (2.0/3.0)*Um/c_s << endl;
	cout << "Check 3. BGK Stability. [If tau < 0.55, tau > 0.5 + 0.125*Uavg]" << endl;
	cout << "tau = " << tau << " > " << 0.5 + 0.125*(2.0 / 3.0)*Um << endl;
	cout << "// ======================================================== //" << endl;

	//intitalize variables
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {

			Ux[i + nx*j] = 0.0;
			Uy[i + nx*j] = 0.0;
			U[i + nx*j] = 0.0;
			UxN[i + nx*j] = 0.0;
			UyN[i + nx*j] = 0.0;
			UN[i + nx*j] = 0.0;
			P[i + nx*j] = 0.0;

			for (k = 0; k < a; k++) {
				ftemp[i + nx*j + nx*ny*k] = 0.0;
				feq[i + nx*j + nx*ny*k] = 0.0;
				fN[i + nx*j + nx*ny*k] = 0.0;
			}

			if (!is_solid_node[i + nx*j]) rho[i + nx*j] = 1.0;
			else rho[i + nx*j] = 1.0;

			f[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j];
			f[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j];
		}
	}

	hipMemcpy(d_rho, rho, nx*ny * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, nx*ny*a * sizeof(float), hipMemcpyHostToDevice);



	//set velocity profile at inlet
	for (j = 0; j < ny; j++) {
		Ux0_p[j] = 4.0*Um_p / (pow(Ly, 2))*(((float)j + 1) - 0.5)*del_y_p*(Ly - (((float)j + 1) - 0.5)*del_y_p);
//		Ux0[j] = 4.0*Um / (pow(ny, 2))*(((float)j + 1) - 0.5)*del_y*(ny - (((float)j + 1) - 0.5)*del_y);
		Ux0[j] = Ux0_p[j] * (del_t_p / del_y_p);

		fout_GPU_Ux0 << Ux0[j] << endl;
	}
	hipMemcpy(d_Ux0, Ux0, ny * sizeof(float), hipMemcpyHostToDevice);
// ============================================================================ //
}

__global__ 
void Kernel_Streaming(float* f, float* ftemp, int* is_boundary_node, int* is_solid_node, int nx, int ny, int a, float ic, float jc, float r) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	int in, ip, jn, jp;

	in = i - 1;
	ip = i + 1;
	jn = j - 1;
	jp = j + 1;

	float dist = sqrt(pow((float)i - ic, 2) + pow((float)j - jc, 2)); 
	float q = dist - r;


	if (!is_boundary_node[i + nx*j]) {
		if (!is_solid_node[i + nx*j]) {
			
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];


			if (!is_solid_node[ip + nx*j]) ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 3] = 2.0 * q * f[i + nx*j + nx*ny * 1] + (1.0 - 2.0*q)*f[(i - 1) + nx*j + nx*ny * 1];
				else ftemp[i + nx*j + nx*ny * 3] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 1] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 3];
			}

			if (!is_solid_node[i + nx*jp]) ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 4] = 2.0 * q * f[i + nx*j + nx*ny * 2] + (1.0 - 2.0*q)*f[i + nx*(j - 1) + nx*ny * 2];
				else ftemp[i + nx*j + nx*ny * 4] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 2] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 4];
			}

			if (!is_solid_node[in + nx*j]) ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 1] = 2.0 * q * f[i + nx*j + nx*ny * 3] + (1.0 - 2.0*q)*f[(i + 1) + nx*j + nx*ny * 3];
				else ftemp[i + nx*j + nx*ny * 1] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 3] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 1];
			}

			if (!is_solid_node[i + nx*jn]) ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 2] = 2.0 * q * f[i + nx*j + nx*ny * 4] + (1.0 - 2.0*q)*f[i + nx*(j + 1) + nx*ny * 4];
				else ftemp[i + nx*j + nx*ny * 2] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 4] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 2];
			}

			if (!is_solid_node[ip + nx*jp]) ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 7] = 2.0 * q * f[i + nx*j + nx*ny * 5] + (1.0 - 2.0*q)*f[(i - 1) + nx*(j - 1) + nx*ny * 5];
				else ftemp[i + nx*j + nx*ny * 7] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 5] + (2.0*q - 1) / (2.0*q)*f[i + nx*j + nx*ny * 7];
			}

			if (!is_solid_node[in + nx*jp]) ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 8] = 2.0 * q * f[i + nx*j + nx*ny * 6] + (1.0 - 2.0*q)*f[(i + 1) + nx*(j - 1) + nx*ny * 6];
				else ftemp[i + nx*j + nx*ny * 8] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 6] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 8];
			}

			if (!is_solid_node[in + nx*jn]) ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 5] = 2.0 * q * f[i + nx*j + nx*ny * 7] + (1.0 - 2.0*q)*f[(i + 1) + nx*(j + 1) + nx*ny * 7];
				else ftemp[i + nx*j + nx*ny * 5] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 7] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 5];
			}

			if (!is_solid_node[ip + nx*jn]) ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
			else {
				if (q < 0.5) ftemp[i + nx*j + nx*ny * 6] = 2.0 * q * f[i + nx*j + nx*ny * 8] + (1.0 - 2.0*q)*f[(i - 1) + nx*(j + 1) + nx*ny * 8];
				else ftemp[i + nx*j + nx*ny * 6] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 8] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 6];
			}

		}
	}
	else {
		if ((i == 0) && (j > 0 && j < ny - 1)) {				//INLET
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
			ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
			ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
			ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
			ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
		}
		else if ((i > 0 && i < nx - 1) && (j == ny - 1)) {			//TOP
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
			ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
			ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
			ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
			ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
		}
		else if ((i > 0 && i < nx - 1) && (j == 0)) {				//BOTTOM		
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
			ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
			ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
			ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
			ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
		}
		else if ((i == nx - 1) && (j > 0 && j < ny - 1)) {			//OUTLET
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
			ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
			ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
			ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
			ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
		}
		else if ((i == 0) && (j == 0)) {							//BOTTOM-LEFT
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
			ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
			ftemp[ip + nx*jp + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
		}
		else if ((i == 0) && (j == ny - 1)) {						//TOP-LEFT
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[ip + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 1];
			ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
			ftemp[ip + nx*jn + nx*ny * 8] = f[i + nx*j + nx*ny * 8];
		}
		else if ((i == nx - 1) && (j == ny - 1)) {					//TOP-RIGHT			
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
			ftemp[i + nx*jn + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
			ftemp[in + nx*jn + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
		}
		else if ((i == nx - 1) && (j == 0)) {						//BOTTOM-RIGHT	
			ftemp[i + nx*j + nx*ny * 0] = f[i + nx*j + nx*ny * 0];
			ftemp[i + nx*jp + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
			ftemp[in + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 3];
			ftemp[in + nx*jp + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
		}
	}

}
void LBM_GPU::Streaming() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Streaming << < dimGrid, dimBlock >> > (d_f, d_ftemp, d_is_boundary_node, d_is_solid_node, nx, ny, a, ic, jc, r);
}

__global__ 
void Kernel_BC_bounceback(float* f, float* ftemp, float* rho, float* Ux, float* Uy, float* Ux0, float rho1, int nx, int ny, int a) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	float rho0, ru, Ux1, Uy1, rho_extra, Ux_extra, Uy_extra;
// ============================================================================ //
//  TOP BOUNDARY (HALF-AWAY BOUNCEBACK)
// ============================================================================ //
	if ((i > 0 && i < nx - 1) && (j == ny - 1)){

		//Bounce-back boundary
		ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];

		//Periodic boundary
		/*ftemp[i + nx*0 + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[(i + 1) + nx*0 + nx*ny * 5] = f[i + nx*j + nx*ny * 5];
		ftemp[(i - 1) + nx*0 + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
*/

		//Velocity boundary(first order)
		/*rho_extra = rho[i + nx*(j - 1)] + 0.5 * (rho[i + nx*(j - 1)] - rho[i + nx*(j - 2)]);
		Ux_extra = Ux[i + nx*(j - 1)];
		ru = rho_extra*Ux_extra;

		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;*/


		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[i + nx*(j - 1) + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*(j - 1) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*(j - 1) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*(j - 1) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*(j - 1) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*(j - 1) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*(j - 1) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*(j - 1) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*(j - 1) + nx*ny * 8];
*/

		//Extrapolation high order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[i + nx*(j - 1) + nx*ny * 0] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 0] - ftemp[i + nx*(j - 2) + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*(j - 1) + nx*ny * 1] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 1] - ftemp[i + nx*(j - 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*(j - 1) + nx*ny * 2] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 2] - ftemp[i + nx*(j - 2) + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*(j - 1) + nx*ny * 3] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 3] - ftemp[i + nx*(j - 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*(j - 1) + nx*ny * 4] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 4] - ftemp[i + nx*(j - 2) + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*(j - 1) + nx*ny * 5] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 5] - ftemp[i + nx*(j - 2) + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*(j - 1) + nx*ny * 6] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 6] - ftemp[i + nx*(j - 2) + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*(j - 1) + nx*ny * 7] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 7] - ftemp[i + nx*(j - 2) + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*(j - 1) + nx*ny * 8] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 8] - ftemp[i + nx*(j - 2) + nx*ny * 8]);
		*/

		//Extrapolation 2nd order
		/*ftemp[i + nx*j + nx*ny * 4] = 2.0 * ftemp[i + nx*(j - 1) + nx*ny * 4] - ftemp[i + nx*(j - 2) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 7] = 2.0 * ftemp[i + nx*(j - 1) + nx*ny * 7] - ftemp[i + nx*(j - 2) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = 2.0 * ftemp[i + nx*(j - 1) + nx*ny * 8] - ftemp[i + nx*(j - 2) + nx*ny * 8];
	*/


		//Equilibrium
		/*float c = 1;
		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
*/

		//NEBB method
		/*ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] + (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 1] - ftemp[i + nx*j + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] - (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 1] - ftemp[i + nx*j + nx*ny * 3]);
		*/

	}
// ============================================================================ //


// ============================================================================ //
//	BOTTOM BOUNDARY (HALF-AWAY BOUNCEBACK)
// ============================================================================ //
	if ((i > 0 && i < nx - 1) && (j == 0)){

		//Bounce-back boundary
		ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];


		//Periodic boundary
		/*ftemp[i + nx*(ny - 1) + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[(i - 1) + nx*(ny - 1) + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
		ftemp[(i + 1) + nx*(ny - 1) + nx*ny * 8] = f[i + nx*j + nx*ny * 8];*/


		//Velocity boundary(first order)
		/*rho_extra = rho[i + nx*(j + 1)] + 0.5 * (rho[i + nx*(j + 1)] - rho[i + nx*(j + 2)]);
		Ux_extra = Ux[i + nx*(j + 1)];
		ru = rho_extra*Ux_extra;

		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 6.0)*ru;*/


		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[i + nx*(j + 1) + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*(j + 1) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*(j + 1) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*(j + 1) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*(j + 1) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*(j + 1) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*(j + 1) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*(j + 1) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*(j + 1) + nx*ny * 8];
	*/

		//Extrapolation high order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[i + nx*(j + 1) + nx*ny * 0] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 0] - ftemp[i + nx*(j + 2) + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*(j + 1) + nx*ny * 1] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 1] - ftemp[i + nx*(j + 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*(j + 1) + nx*ny * 2] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 2] - ftemp[i + nx*(j + 2) + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*(j + 1) + nx*ny * 3] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 3] - ftemp[i + nx*(j + 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*(j + 1) + nx*ny * 4] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 4] - ftemp[i + nx*(j + 2) + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*(j + 1) + nx*ny * 5] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 5] - ftemp[i + nx*(j + 2) + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*(j + 1) + nx*ny * 6] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 6] - ftemp[i + nx*(j + 2) + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*(j + 1) + nx*ny * 7] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 7] - ftemp[i + nx*(j + 2) + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*(j + 1) + nx*ny * 8] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 8] - ftemp[i + nx*(j + 2) + nx*ny * 8]);
*/

		//Extrapolation 2nd order
		/*ftemp[i + nx*j + nx*ny * 2] = 2.0 * ftemp[i + nx*(j + 1) + nx*ny * 2] - ftemp[i + nx*(j + 2) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 5] = 2.0 * ftemp[i + nx*(j + 1) + nx*ny * 5] - ftemp[i + nx*(j + 2) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = 2.0 * ftemp[i + nx*(j + 1) + nx*ny * 6] - ftemp[i + nx*(j + 2) + nx*ny * 6];
	*/


		//Equilibrium
		/*float c = 1;
		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	*/

		//NEBB method
		/*ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 3] - ftemp[i + nx*j + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 3] - ftemp[i + nx*j + nx*ny * 1]);
*/

	}
// ============================================================================ //


// ============================================================================ //
//	LEFT BOUNDARY (VELOCITY)
// ============================================================================ //
	if ((i == 0) && (j > 0 && j < ny - 1)) {
		/*rho0 = rho[(i + 1) + nx*j] + 0.5*(rho[(i + 1) + nx*j] - rho[(i + 2) + nx*j]);
		ru = rho0 * Ux0;

		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;*/


		/*rho0 = ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 4]
			+ 2.0*(ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 6]);
		

		ru = rho0 * Ux0[j];

		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;*/




		//Zou - He boundary
		rho0 = (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 4]
			+ 2.0*(ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7])) / (1.0 - Ux0[j]);
		ru = rho0 * Ux0[j];

		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru - (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 2] - ftemp[i + nx*j + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru + (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 2] - ftemp[i + nx*j + nx*ny * 4]);


		//wet-node method
//		rho0 = (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 4]
//			+ 2.0*(ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7])) / (1.0 - Ux0[j]);
//		ru = rho0 * Ux0[j];
////		ru = rho0 * 0.06;
//
//		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
//		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru - (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 2] - ftemp[i + nx*j + nx*ny * 4]);
//		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru + (1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 2] - ftemp[i + nx*j + nx*ny * 4]);
		
		
		//Equilibrium
		/*float c = 1;
		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux0[j] + (4.5 / pow(c, 4)) * pow(Ux0[j], 2) - (1.5 / pow(c, 2)) * (pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux0[j] + (4.5 / pow(c, 4))*pow(Ux0[j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux0[j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux0[j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux0[j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux0[j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux0[j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux0[j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux0[j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux0[j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
	*/
	}
// ============================================================================ //


// ============================================================================ //
//  RIGHT BOUNDARY (EXTRAPOLATION)
// ============================================================================ //
	if ((i == nx - 1) && (j > 0 && j < ny - 1)) {

		//Extrapolation
//		ftemp[i + nx*j + nx*ny * 0] = ftemp[(i - 1) + nx*j + nx*ny * 0] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 0] - ftemp[(i - 2) + nx*j + nx*ny * 0]);
//		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i - 1) + nx*j + nx*ny * 1] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 1] - ftemp[(i - 2) + nx*j + nx*ny * 1]);
//		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i - 1) + nx*j + nx*ny * 2] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 2] - ftemp[(i - 2) + nx*j + nx*ny * 2]);
//		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*j + nx*ny * 3] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 3] - ftemp[(i - 2) + nx*j + nx*ny * 3]);
//		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i - 1) + nx*j + nx*ny * 4] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 4] - ftemp[(i - 2) + nx*j + nx*ny * 4]);
//		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i - 1) + nx*j + nx*ny * 5] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 5] - ftemp[(i - 2) + nx*j + nx*ny * 5]);
//		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*j + nx*ny * 6] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 6] - ftemp[(i - 2) + nx*j + nx*ny * 6]);
//		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*j + nx*ny * 7] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 7] - ftemp[(i - 2) + nx*j + nx*ny * 7]);
//		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i - 1) + nx*j + nx*ny * 8] + 0.5 * (ftemp[(i - 1) + nx*j + nx*ny * 8] - ftemp[(i - 2) + nx*j + nx*ny * 8]);
		

		//Extrapolation high order
		/*ftemp[i + nx*j + nx*ny * 0] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 0] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 0] + ftemp[(i - 3) + nx*j + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 1] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 1] + ftemp[(i - 3) + nx*j + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 2] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 2] + ftemp[(i - 3) + nx*j + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 3] + ftemp[(i - 3) + nx*j + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 4] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 4] + ftemp[(i - 3) + nx*j + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 5] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 5] + ftemp[(i - 3) + nx*j + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 6] + ftemp[(i - 3) + nx*j + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 7] + ftemp[(i - 3) + nx*j + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 8] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 8] + ftemp[(i - 3) + nx*j + nx*ny * 8]);
	*/

		//Extrapolation type2
		/*ftemp[i + nx*j + nx*ny * 3] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - ftemp[(i - 2) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - ftemp[(i - 2) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - ftemp[(i - 2) + nx*j + nx*ny * 7];*/


		//Extrapolation first order
		ftemp[i + nx*j + nx*ny * 0] = ftemp[(i - 1) + nx*j + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i - 1) + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i - 1) + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i - 1) + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i - 1) + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i - 1) + nx*j + nx*ny * 8];


		//Extrapolation second order
		/*ftemp[i + nx*j + nx*ny * 0] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 0] - ftemp[(i - 2) + nx*j + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 1] - ftemp[(i - 2) + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 2] - ftemp[(i - 2) + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - ftemp[(i - 2) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 4] - ftemp[(i - 2) + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 5] - ftemp[(i - 2) + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - ftemp[(i - 2) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - ftemp[(i - 2) + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 8] - ftemp[(i - 2) + nx*j + nx*ny * 8];*/



		//Velocity boundary (first order)
		/*rho_extra = rho[(i - 1) + nx*j] + 0.5*(rho[(i - 1) + nx*j] - rho[(i - 2) + nx*j]);
		Ux_extra = Ux[(i - 1) + nx*j];
		Uy_extra = Uy[(i - 1) + nx*j];

		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1] - (2.0 / 3.0)*rho_extra*Ux_extra;
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 6.0)*rho_extra*(Ux_extra - Uy_extra);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 6.0)*rho_extra*(Ux_extra + Uy_extra);*/


		//Pressure boundary
		/*Ux1 = Ux[(i - 1) + nx*j] + 0.5*(Ux[(i - 1) + nx*j] - Ux[(i - 2) + nx*j]);
		Uy1 = Uy[(i - 1) + nx*j] + 0.5*(Uy[(i - 1) + nx*j] - Uy[(i - 2) + nx*j]);

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 - Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 + Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
*/

		//wet-node method
		/*Ux1 = -1.0 + (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 4]
			+ 2.0*(ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 8])) / rho1;
		ru = rho1 * Ux1;

		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1] - (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 6.0)*ru -(1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 2] - ftemp[i + nx*j + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 6.0)*ru +(1.0 / 2.0)*(ftemp[i + nx*j + nx*ny * 2] - ftemp[i + nx*j + nx*ny * 4]);
*/
	}
// ============================================================================ //


// ============================================================================ //
//	TOP-LEFT CORNER (EQUILIBRIUM)
// ============================================================================ //
	if ((i == 0) && (j == ny - 1)) {
		

		//case 1
	//	ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
	//	ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];

	////	rho0 = rho[(i + 1) + nx*(j - 1)] + 0.5*(rho[(i + 1) + nx*(j - 1)] - rho[(i + 2) + nx*(j - 2)]);
	//	rho0 = rho[i + nx*(j - 1)];
	//	ru = rho0 * Ux0[j];

	//	ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
	//	ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
	//	ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;




		//case 2
		//ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		//ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
		//ftemp[i + nx*j + nx*ny * 5] = -ftemp[i + nx*j + nx*ny * 7];

		//rho0 = rho[(i + 1) + nx*(j - 1)] + 0.5*(rho[(i + 1) + nx*(j - 1)] - rho[(i + 2) + nx*(j - 2)]);
		//ru = rho0 * Ux0;

		//ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		//ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;

		//ftemp[i + nx*j + nx*ny * 0] = rho0 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
		//	+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);


		//case 3
		/*ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];*/

	

		//case 4
		/*ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];

		rho0 = rho[(i + 0) + nx*(j - 1)] + 0.5*(rho[(i + 0) + nx*(j - 1)] - rho[(i + 0) + nx*(j - 2)]);
		ru = rho0 * Ux0[j];

		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		*/
	
		//case 5
////		rho0 = rho[(i + 0) + nx*(j - 1)] + 0.5*(rho[(i + 0) + nx*(j - 1)] - rho[(i + 0) + nx*(j - 2)]);
//		rho0 = rho[i + nx*(j - 1)];
////		rho0 = 1.0;
//		ru = rho0 * Ux0[j];
////		ru = rho0* 0.005;
//
//		ftemp[i + nx*j + nx*ny * 7] = -(1.0 / 12.0) * ru;
//		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 12.0) * ru;
//
//		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
//		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
//		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;
//
//		ftemp[i + nx*j + nx*ny * 0] = rho0 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
//			+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);


		//Periodic + Velocity
		/*ftemp[i + nx*0 + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[(i + 1) + nx*0 + nx*ny * 5] = f[i + nx*j + nx*ny * 5];

		rho0 = rho[(i + 1) + nx*(j - 1)] + 0.5*(rho[(i + 1) + nx*(j - 1)] - rho[(i + 2) + nx*(j - 2)]);
		ru = rho0 * Ux0;

		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;*/


		//wet-node method
//		rho0 = rho[i + nx*(j - 1)];
////		rho0 = 1.001;
////		rho0 = rho[(i + 0) + nx*(j - 1)] + 0.5*(rho[(i + 0) + nx*(j - 1)] - rho[(i + 0) + nx*(j - 2)]);
////		rho0 = rho1;
//
//		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3];
//		ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
//		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
//		ftemp[i + nx*j + nx*ny * 5] = 0.5 * (rho0 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
//			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 8]));
//		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];


		//Equilibrium
		float c = 1;
		/*ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux0[j] + (4.5 / pow(c, 4)) * pow(Ux0[j], 2) - (1.5 / pow(c, 2)) * (pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux0[j] + (4.5 / pow(c, 4))*pow(Ux0[j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux0[j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux0[j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux0[j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux0[j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux0[j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux0[j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux0[j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux0[j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
	*/	

		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));

	}
// ============================================================================ //


// ============================================================================ //
//	BOTTOM-LEFT CORNER (EQUILIBRIUM)
// ============================================================================ //
	if ((i == 0) && (j == 0)) {
		
		//case 1
	//	ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
	//	ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];

	////	rho0 = rho[(i + 1) + nx*(j + 1)] + 0.5*(rho[(i + 1) + nx*(j + 1)] - rho[(i + 2) + nx*(j + 2)]);
	//	rho0 = rho[i + nx*(j + 1)];
	//	ru = rho0 * Ux0[j];

	//	ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
	//	ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
	//	ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;


		//case 2
		//ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		//ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		//ftemp[i + nx*j + nx*ny * 8] = -ftemp[i + nx*j + nx*ny * 6];

		//rho0 = rho[(i + 1) + nx*(j + 1)] + 0.5*(rho[(i + 1) + nx*(j + 1)] - rho[(i + 2) + nx*(j + 2)]);
		//ru = rho0 * Ux0;

		//ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		//ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		//
		//ftemp[i + nx*j + nx*ny * 0] = rho0 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
		//	+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);



		//case 3
		/*ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
*/

		//case 4
		/*ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];

		rho0 = rho[(i + 0) + nx*(j + 1)] + 0.5*(rho[(i + 0) + nx*(j + 1)] - rho[(i + 0) + nx*(j + 2)]);
		ru = rho0 * Ux0[j];

		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;
*/

		//case 5		
////		rho0 = rho[(i + 0) + nx*(j + 1)] + 0.5*(rho[(i + 0) + nx*(j + 1)] - rho[(i + 0) + nx*(j + 2)]);
//		rho0 = rho[i + nx*(j + 1)];
////		rho0 = 1.0;
//		ru = rho0 * Ux0[j];
////		ru = rho0* 0.005;
//
//		ftemp[i + nx*j + nx*ny * 6] = -(1.0 / 12.0) * ru;
//		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 12.0) * ru;
//
//		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
//		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
//		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
//		
//		ftemp[i + nx*j + nx*ny * 0] = rho0 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
//			+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);



		//Periodic + Velocity
		/*ftemp[i + nx*(ny - 1) + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[(i + 1) + nx*(ny - 1) + nx*ny * 8] = f[i + nx*j + nx*ny * 8];

		rho0 = rho[(i + 1) + nx*(j + 1)] + 0.5*(rho[(i + 1) + nx*(j + 1)] - rho[(i + 2) + nx*(j + 2)]);
		ru = rho0 * Ux0;

		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3] + (2.0 / 3.0)*ru;
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;
*/


		//wet-node method
//		rho0 = rho[i + nx*(j + 1)];
////		rho0 = 1.001;
////		rho0 = rho[(i + 0) + nx*(j + 1)] + 0.5*(rho[(i + 0) + nx*(j + 1)] - rho[(i + 0) + nx*(j + 2)]);
////		rho0 = rho1;
//
//		ftemp[i + nx*j + nx*ny * 1] = f[i + nx*j + nx*ny * 3];
//		ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
//		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
//		ftemp[i + nx*j + nx*ny * 6] = 0.5 * (rho0 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
//			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 7]));
//		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];


		//Equilibrium
		float c = 1;
		/*ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux0[j] + (4.5 / pow(c, 4)) * pow(Ux0[j], 2) - (1.5 / pow(c, 2)) * (pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux0[j] + (4.5 / pow(c, 4))*pow(Ux0[j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux0[j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux0[j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux0[j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux0[j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux0[j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux0[j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux0[j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux0[j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux0[j], 2) + pow(Uy[i + nx*j], 2)));
*/

		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));

	}
// ============================================================================ //

	
// ============================================================================ //
//	TOP-RIGHT CORNER (EQUILIBRIUM)
// ============================================================================ //
	if ((i == nx - 1) && (j == ny - 1)) {

		//case 1
		/*ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];

		Ux1 = Ux[(i - 1) + nx*(j - 1)] + 0.5*(Ux[(i - 1) + nx*(j - 1)] - Ux[(i - 2) + nx*(j - 2)]);
		Uy1 = Uy[(i - 1) + nx*(j - 1)] + 0.5*(Uy[(i - 1) + nx*(j - 1)] - Uy[(i - 2) + nx*(j - 2)]);

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 - Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 + Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));*/


		

		//case 2
		//ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		//ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
		//ftemp[i + nx*j + nx*ny * 6] = -ftemp[i + nx*j + nx*ny * 8];

		//Ux1 = Ux[(i - 1) + nx*(j - 1)] + 0.5*(Ux[(i - 1) + nx*(j - 1)] - Ux[(i - 2) + nx*(j - 2)]);

		//ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
		//ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);

		//ftemp[i + nx*j + nx*ny * 0] = rho1 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
		//	+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);


		//case 3
		/*ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];*/


		//case 4
		/*ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 8] = f[i + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];

		Ux1 = Ux[(i - 1) + nx*(j - 1)] + 0.5*(Ux[(i - 1) + nx*(j - 1)] - Ux[(i - 2) + nx*(j - 2)]);

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
		ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
		*/


		//case 5
//		Ux1 = Ux[(i - 1) + nx*(j - 0)];
////		rho_extra = rho[(i - 1) + nx*(j - 1)] + 0.5*(rho[(i - 1) + nx*(j - 1)] - rho[(i - 2) + nx*(j - 2)]);
//		rho_extra = rho[(i - 1) + nx*(j - 0)];
//		ru = rho_extra * Ux1;
//		
//		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 12.0) * ru;
//		ftemp[i + nx*j + nx*ny * 6] = -(1.0 / 12.0) * ru;
//
//		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
//		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1] - (2.0 / 3.0) * ru;
//		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 6.0) * ru;
//
//		ftemp[i + nx*j + nx*ny * 0] = rho1 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
//			+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);


		//Periodic + Bounce back
		/*ftemp[i + nx*0 + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[(i - 1) + nx*0 + nx*ny * 6] = f[i + nx*j + nx*ny * 6];

		Ux1 = Ux[(i - 1) + nx*(j - 1)];
		Uy1 = Uy[(i - 1) + nx*(j - 1)];

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 - Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 + Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
*/


		//Periodic + Extrapolation
		/*ftemp[i + nx * 0 + nx*ny * 2] = f[i + nx*j + nx*ny * 2];
		ftemp[(i - 1) + nx * 0 + nx*ny * 6] = f[i + nx*j + nx*ny * 6];
*/


		//wet-node method
////		rho0 = rho[(i - 1) + nx*(j - 0)];
//		rho0 = rho1;
////		rho0 = rho[(i - 0) + nx*(j - 1)] + 0.5*(rho[(i - 0) + nx*(j - 1)] - rho[(i - 0) + nx*(j - 2)]);
//
//		ftemp[i + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 1];
//		ftemp[i + nx*j + nx*ny * 4] = f[i + nx*j + nx*ny * 2];
//		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];
//		ftemp[i + nx*j + nx*ny * 6] = 0.5 * (rho0 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
//			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 7]));
//		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];


		//Equilibrium
		float c = 1;
		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));

	}
// ============================================================================ //


// ============================================================================ //
//	BOTTOM-RIGHT CORNER (EQUILIBRIUM)
// ============================================================================ //
	if ((i == nx - 1) && (j == 0)) {

		//case 1
		/*ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];

		Ux1 = Ux[(i - 1) + nx*(j + 1)] + 0.5*(Ux[(i - 1) + nx*(j + 1)] - Ux[(i - 2) + nx*(j + 2)]);
		Uy1 = Uy[(i - 1) + nx*(j + 1)] + 0.5*(Uy[(i - 1) + nx*(j + 1)] - Uy[(i - 2) + nx*(j + 2)]);

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 - Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 + Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
*/

		

		//case 2
		//ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		//ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		//ftemp[i + nx*j + nx*ny * 7] = -ftemp[i + nx*j + nx*ny * 5];

		//Ux1 = Ux[(i - 1) + nx*(j + 1)] + 0.5*(Ux[(i - 1) + nx*(j + 1)] - Ux[(i - 2) + nx*(j + 2)]);

		//ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
		//ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
		//
		//ftemp[i + nx*j + nx*ny * 0] = rho1 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
		//	+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);


		//case 3
		/*ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 7] = f[i + nx*j + nx*ny * 5];*/


		//case 4
		/*ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = f[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];

		Ux1 = Ux[(i - 1) + nx*(j + 1)] + 0.5*(Ux[(i - 1) + nx*(j + 1)] - Ux[(i - 2) + nx*(j + 2)]);

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
		ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + 3.0 * Ux1*Ux1);
*/

		//case 5
//		Ux1 = Ux[(i - 1) + nx*(j + 0)];
////		rho_extra = rho[(i - 1) + nx*(j + 1)] + 0.5*(rho[(i - 1) + nx*(j + 1)] - rho[(i - 2) + nx*(j + 2)]);
//		rho_extra = rho[(i - 1) + nx*(j + 0)];
//
//		ru = Ux1 * rho_extra;
//
//		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 12.0) * ru;
//		ftemp[i + nx*j + nx*ny * 7] = -(1.0 / 12.0) * ru;
//
//		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
//		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1] - (2.0 / 3.0) * ru;
//		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 6.0) * ru;
//		
//		ftemp[i + nx*j + nx*ny * 0] = rho1 - (ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
//			+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7] + ftemp[i + nx*j + nx*ny * 8]);
//		
		
		//Periodic + Bounce back
		/*ftemp[i + nx*(ny - 1) + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[(i - 1) + nx*(ny - 1) + nx*ny * 7] = f[i + nx*j + nx*ny * 7];

		Ux1 = Ux[(i - 1) + nx*(j + 1)];
		Uy1 = Uy[(i - 1) + nx*(j + 1)];

		ftemp[i + nx*j + nx*ny * 3] = -f[i + nx*j + nx*ny * 1] + (2.0 / 9.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 6] = -f[i + nx*j + nx*ny * 8] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 - Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));
		ftemp[i + nx*j + nx*ny * 7] = -f[i + nx*j + nx*ny * 5] + (1.0 / 18.0) * rho1 * (1.0 + (9.0 / 2.0)*pow(Ux1 + Uy1, 2) - (3.0 / 2.0)*(Ux1*Ux1 + Uy1*Uy1));*/


		//Periodic + Extrapolation
		/*ftemp[i + nx*(ny - 1) + nx*ny * 4] = f[i + nx*j + nx*ny * 4];
		ftemp[(i - 1) + nx*(ny - 1) + nx*ny * 7] = f[i + nx*j + nx*ny * 7];
*/

		//wet-node method
	////	rho0 = rho[(i - 1) + nx*(j + 0)];
	//	rho0 = rho1;
	////	rho0 = rho[(i - 0) + nx*(j + 1)] + 0.5*(rho[(i - 0) + nx*(j + 1)] - rho[(i - 0) + nx*(j + 2)]);

	//	ftemp[i + nx*j + nx*ny * 2] = f[i + nx*j + nx*ny * 4];
	//	ftemp[i + nx*j + nx*ny * 6] = f[i + nx*j + nx*ny * 8];
	//	ftemp[i + nx*j + nx*ny * 3] = f[i + nx*j + nx*ny * 1];
	//	ftemp[i + nx*j + nx*ny * 5] = 0.5 * (rho0 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
	//		+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 8]));
	//	ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];



		//Equilibrium
		float c = 1;
		ftemp[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
		ftemp[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));

	}
// ============================================================================ //
}
void LBM_GPU::BC_bounceback() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_BC_bounceback << < dimGrid, dimBlock >> > (d_f, d_ftemp, d_rho, d_Ux, d_Uy, d_Ux0, rho1, nx, ny, a);
}

__global__
void Kernel_BC_extra(float* ftemp, float* Ux, float* rho, int nx, int ny, int a, float rho1) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	float ru, Ux_extra, rho_extra;
// ============================================================================ //
//	TOP-LEFT CORNER (VELOCITY & PERIODIC)
// ============================================================================ //
	if ((i == 0) && (j == ny - 1)) {
		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i + 0) + nx*(j - 1) + nx*ny * 8];*/
		

		//Extrapolation high order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 0] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 0] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 1] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 1] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 2] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 2] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 3] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 3] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 4] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 4] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 5] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 5] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 6] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 6] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 7] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 7] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i + 1) + nx*(j - 1) + nx*ny * 8] + 0.5 * (ftemp[(i + 1) + nx*(j - 1) + nx*ny * 8] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 8]);
*/

		//Extrapolation 2nd order + moving wall
		/*ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*(j - 1) + nx*ny * 1] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 1] - ftemp[i + nx*(j - 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*(j - 1) + nx*ny * 5] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 5] - ftemp[i + nx*(j - 2) + nx*ny * 5]);
	
		rho_extra = rho[i + nx*(j - 1)] + 0.5 * (rho[i + nx*(j - 1)] - rho[i + nx*(j - 2)]);
		Ux_extra = Ux[i + nx*(j - 1)] + 0.5 * (Ux[i + nx*(j - 1)] - Ux[i + nx*(j - 2)]);
		ru = rho_extra*Ux_extra;

		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;
*/

		//Extrapolation 2nd order
		/*ftemp[i + nx*j + nx*ny * 1] = 2.0*ftemp[(i + 1) + nx*(j - 1) + nx*ny * 1] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 5] = 2.0*ftemp[(i + 1) + nx*(j - 1) + nx*ny * 5] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 4] = 2.0*ftemp[(i + 1) + nx*(j - 1) + nx*ny * 4] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i + 1) + nx*(j - 1) + nx*ny * 7] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = 2.0*ftemp[(i + 1) + nx*(j - 1) + nx*ny * 8] - ftemp[(i + 2) + nx*(j - 2) + nx*ny * 8];
	*/

		//Zou - He boundary
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 5] = 0.5 * (rho1 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 8]));
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];
	}
// ============================================================================ //


// ============================================================================ //
//	BOTTOM-LEFT CORNER (VELOCITY & PERIODIC)
// ============================================================================ //
	if ((i == 0) && (j == 0)) {
		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i + 0) + nx*(j + 1) + nx*ny * 8];*/

		//Extrapolation high order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 0] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 0] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 1] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 1] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 2] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 2] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 3] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 3] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 4] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 4] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 5] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 5] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 6] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 6] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 7] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 7] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i + 1) + nx*(j + 1) + nx*ny * 8] + 0.5 * (ftemp[(i + 1) + nx*(j + 1) + nx*ny * 8] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 8]);
*/

		//Extrapolation 2nd order + moving wall
		/*ftemp[i + nx*j + nx*ny * 1] = ftemp[i+ nx*(j + 1) + nx*ny * 1] + 0.5 * (ftemp[i+ nx*(j + 1) + nx*ny * 1] - ftemp[i+ nx*(j + 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i+ nx*(j + 1) + nx*ny * 8] + 0.5 * (ftemp[i+ nx*(j + 1) + nx*ny * 8] - ftemp[i+ nx*(j + 2) + nx*ny * 8]);

		rho_extra = rho[i + nx*(j + 1)] + 0.5 * (rho[i + nx*(j + 1)] - rho[i + nx*(j + 2)]);
		Ux_extra = Ux[i + nx*(j + 1)] + 0.5 * (Ux[i + nx*(j + 1)] - Ux[i + nx*(j + 2)]);
		ru = rho_extra*Ux_extra;

		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 6.0)*ru;
*/
		//Extrapolation 2nd order
		/*ftemp[i + nx*j + nx*ny * 1] = 2.0*ftemp[(i + 1) + nx*(j + 1) + nx*ny * 1] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 8] = 2.0*ftemp[(i + 1) + nx*(j + 1) + nx*ny * 8] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 2] = 2.0*ftemp[(i + 1) + nx*(j + 1) + nx*ny * 2] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 5] = 2.0*ftemp[(i + 1) + nx*(j + 1) + nx*ny * 5] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i + 1) + nx*(j + 1) + nx*ny * 6] - ftemp[(i + 2) + nx*(j + 2) + nx*ny * 6];
*/

		//Zou - He boundary
		ftemp[i + nx*j + nx*ny * 1] = ftemp[i + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 6] = 0.5 * (rho1 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 7]));
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];
	}
// ============================================================================ //


// ============================================================================ //
//	TOP-RIGHT CORNER (EXTRAPOLATION & PERIODIC)
// ============================================================================ //
	if ((i == nx - 1) && (j == ny - 1)) {

		//Extrapolation
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 0] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 0] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 1] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 1] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 2] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 2] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 3] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 3] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 4] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 4] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 5] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 5] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 6] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 6] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 7] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 7] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i - 1) + nx*(j - 1) + nx*ny * 8] + 0.5 * (ftemp[(i - 1) + nx*(j - 1) + nx*ny * 8] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 8]);*/

		//Extrapolation high order
		/*ftemp[i + nx*j + nx*ny * 3] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 3] + ftemp[(i - 3) + nx*j + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 6] + ftemp[(i - 3) + nx*j + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 7] + ftemp[(i - 3) + nx*j + nx*ny * 7]);
*/
		//Extrapolation 2nd order
		/*ftemp[i + nx*j + nx*ny * 3] = 2.0*ftemp[(i - 1) + nx*(j - 1) + nx*ny * 3] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i - 1) + nx*(j - 1) + nx*ny * 6] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i - 1) + nx*(j - 1) + nx*ny * 7] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 4] = 2.0*ftemp[(i - 1) + nx*(j - 1) + nx*ny * 4] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 8] = 2.0*ftemp[(i - 1) + nx*(j - 1) + nx*ny * 8] - ftemp[(i - 2) + nx*(j - 2) + nx*ny * 8];
	*/

		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*j + nx*ny * 7];
*/

		//Extrapolation second order
		/*ftemp[i + nx*j + nx*ny * 3] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - ftemp[(i - 2) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - ftemp[(i - 2) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - ftemp[(i - 2) + nx*j + nx*ny * 7];*/

		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i - 1) + nx*(j - 0) + nx*ny * 8];*/


		//Extrapolation 2nd order + moving wall
		/*ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*(j - 1) + nx*ny * 3] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 3] - ftemp[i + nx*(j - 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*(j - 1) + nx*ny * 6] + 0.5 * (ftemp[i + nx*(j - 1) + nx*ny * 6] - ftemp[i + nx*(j - 2) + nx*ny * 6]);

		rho_extra = rho[i + nx*(j - 1)] + 0.5 * (rho[i + nx*(j - 1)] - rho[i + nx*(j - 2)]);
		Ux_extra = Ux[i + nx*(j - 1)] + 0.5 * (Ux[i + nx*(j - 1)] - Ux[i + nx*(j - 2)]);
		ru = rho_extra*Ux_extra;

		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5] - (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6] + (1.0 / 6.0)*ru;
*/

		//Zou - He boundary
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[i + nx*j + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = 0.5 * (rho1 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 7]));
		ftemp[i + nx*j + nx*ny * 8] = ftemp[i + nx*j + nx*ny * 6];
	}
// ============================================================================ //


// ============================================================================ //
//	BOTTOM-RIGHT CORNER (EXTRAPOLATION & PERIODIC)
// ============================================================================ //
	if ((i == nx - 1) && (j == 0)) {

		//Extrapolation
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 0] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 0] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 0]);
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 1] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 1] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 1]);
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 2] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 2] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 2]);
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 3] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 3] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 4] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 4] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 4]);
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 5] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 5] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 5]);
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 6] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 6] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 7] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 7] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 7]);
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i - 1) + nx*(j + 1) + nx*ny * 8] + 0.5 * (ftemp[(i - 1) + nx*(j + 1) + nx*ny * 8] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 8]);
*/

		//Extrapolation high order
	/*	ftemp[i + nx*j + nx*ny * 3] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 3] + ftemp[(i - 3) + nx*j + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 6] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 6] + ftemp[(i - 3) + nx*j + nx*ny * 6]);
		ftemp[i + nx*j + nx*ny * 7] = (1.0 / 3.0) * (7.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - 5.0*ftemp[(i - 2) + nx*j + nx*ny * 7] + ftemp[(i - 3) + nx*j + nx*ny * 7]);
*/

		//Extrapolation 2nd order
		/*ftemp[i + nx*j + nx*ny * 3] = 2.0*ftemp[(i - 1) + nx*(j + 1) + nx*ny * 3] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i - 1) + nx*(j + 1) + nx*ny * 6] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i - 1) + nx*(j + 1) + nx*ny * 7] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 2] = 2.0*ftemp[(i - 1) + nx*(j + 1) + nx*ny * 2] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 5] = 2.0*ftemp[(i - 1) + nx*(j + 1) + nx*ny * 5] - ftemp[(i - 2) + nx*(j + 2) + nx*ny * 5];*/



		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*j + nx*ny * 7];*/

		//Extrapolation second order
		/*ftemp[i + nx*j + nx*ny * 3] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 3] - ftemp[(i - 2) + nx*j + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 6] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 6] - ftemp[(i - 2) + nx*j + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = 2.0*ftemp[(i - 1) + nx*j + nx*ny * 7] - ftemp[(i - 2) + nx*j + nx*ny * 7];
*/

		//Extrapolation first order
		/*ftemp[i + nx*j + nx*ny * 0] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 0];
		ftemp[i + nx*j + nx*ny * 1] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 2] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 2];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 3];
		ftemp[i + nx*j + nx*ny * 4] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 5];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 6];
		ftemp[i + nx*j + nx*ny * 7] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 7];
		ftemp[i + nx*j + nx*ny * 8] = ftemp[(i - 1) + nx*(j + 0) + nx*ny * 8];
*/

		//Extrapolation 2nd order + moving wall
		/*ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*(j + 1) + nx*ny * 3] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 3] - ftemp[i + nx*(j + 2) + nx*ny * 3]);
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*(j + 1) + nx*ny * 7] + 0.5 * (ftemp[i + nx*(j + 1) + nx*ny * 7] - ftemp[i + nx*(j + 2) + nx*ny * 7]);

		rho_extra = rho[i + nx*(j + 1)] + 0.5 * (rho[i + nx*(j + 1)] - rho[i + nx*(j + 2)]);
		Ux_extra = Ux[i + nx*(j + 1)] + 0.5 * (Ux[i + nx*(j + 1)] - Ux[i + nx*(j + 2)]);
		ru = rho_extra*Ux_extra;

		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 5] = ftemp[i + nx*j + nx*ny * 7] + (1.0 / 6.0)*ru;
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8] - (1.0 / 6.0)*ru;
*/

		//Zou - He boundary
		ftemp[i + nx*j + nx*ny * 2] = ftemp[i + nx*j + nx*ny * 4];
		ftemp[i + nx*j + nx*ny * 6] = ftemp[i + nx*j + nx*ny * 8];
		ftemp[i + nx*j + nx*ny * 3] = ftemp[i + nx*j + nx*ny * 1];
		ftemp[i + nx*j + nx*ny * 5] = 0.5 * (rho1 - (ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1] + ftemp[i + nx*j + nx*ny * 2]
			+ ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 8]));
		ftemp[i + nx*j + nx*ny * 7] = ftemp[i + nx*j + nx*ny * 5];
	}
// ============================================================================ //
}
void LBM_GPU::BC_extra() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_BC_extra << < dimGrid, dimBlock >> > (d_ftemp, d_Ux, d_rho, nx, ny, a, rho1);
}

__global__ 
void Kernel_Eq(float* ftemp, float* feq, float* Ux, float* Uy, float* rho, float* ex, float* ey, int nx, int ny, int a, int* is_solid_node, float c) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;


	//Calculation of Macroscopic var 
	if (!is_solid_node[i + nx*j]){
	rho[i + nx*j] = ftemp[i + nx*j + nx*ny * 0] + ftemp[i + nx*j + nx*ny * 1]
		+ ftemp[i + nx*j + nx*ny * 2] + ftemp[i + nx*j + nx*ny * 3] + ftemp[i + nx*j + nx*ny * 4]
		+ ftemp[i + nx*j + nx*ny * 5] + ftemp[i + nx*j + nx*ny * 6] + ftemp[i + nx*j + nx*ny * 7]
		+ ftemp[i + nx*j + nx*ny * 8];

	Ux[i + nx*j] = ftemp[i + nx*j + nx*ny * 1] * ex[1] + ftemp[i + nx*j + nx*ny * 3] * ex[3]
		+ ftemp[i + nx*j + nx*ny * 5] * ex[5] + ftemp[i + nx*j + nx*ny * 6] * ex[6] + ftemp[i + nx*j + nx*ny * 7] * ex[7]
		+ ftemp[i + nx*j + nx*ny * 8] * ex[8];

	Uy[i + nx*j] = ftemp[i + nx*j + nx*ny * 2] * ey[2] + ftemp[i + nx*j + nx*ny * 4] * ey[4]
		+ ftemp[i + nx*j + nx*ny * 5] * ey[5] + ftemp[i + nx*j + nx*ny * 6] * ey[6] + ftemp[i + nx*j + nx*ny * 7] * ey[7]
		+ ftemp[i + nx*j + nx*ny * 8] * ey[8];

	Ux[i + nx*j] /= rho[i + nx*j];
	Uy[i + nx*j] /= rho[i + nx*j];



	feq[i + nx*j + nx*ny * 0] = (4.0 / 9.0) * rho[i + nx*j] * (1.0 - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 1] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4)) * pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2)) * (pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 2] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 3] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Ux[i + nx*j] + (4.5 / pow(c, 4))*pow(Ux[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 4] = (1.0 / 9.0) * rho[i + nx*j] * (1.0 - (3.0 / pow(c, 2)) * Uy[i + nx*j] + (4.5 / pow(c, 4))*pow(Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 5] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 6] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] + Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] + Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 7] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (-Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(-Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));
	feq[i + nx*j + nx*ny * 8] = (1.0 / 36.0) * rho[i + nx*j] * (1.0 + (3.0 / pow(c, 2)) * (Ux[i + nx*j] - Uy[i + nx*j]) + (4.5 / pow(c, 4))*pow(Ux[i + nx*j] - Uy[i + nx*j], 2) - (1.5 / pow(c, 2))*(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2)));

	}
}
__global__
void Kernel_Collision(float* fN, float* ftemp, float* feq, int nx, int ny, int a, float tau, int* is_solid_node) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	if (!is_solid_node[i + nx*j]) {
		fN[i + nx*j + nx*ny*k] = ftemp[i + nx*j + nx*ny*k] - (ftemp[i + nx*j + nx*ny*k] - feq[i + nx*j + nx*ny*k]) / tau;
	}
}
void LBM_GPU::Collision() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Eq << < dimGrid, dimBlock >> > (d_ftemp, d_feq, d_Ux, d_Uy, d_rho, d_ex, d_ey, nx, ny, a, d_is_solid_node, c);
	Kernel_Collision << < dimGrid, dimBlock >> > (d_fN, d_ftemp, d_feq, nx, ny, a, tau, d_is_solid_node);
}

__global__ 
void Kernel_Error(float* f, float* Ux, float* Uy, float* U, float* rho, float* fN, float* UxN, float* UyN, float* UN, float* rhoN, float* ex, float* ey, int nx, int ny, int a, int* is_solid_node) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	if (!is_solid_node[i + nx*j]) {
		rho[i + nx*j] = f[i + nx*j + nx*ny * 0] + f[i + nx*j + nx*ny * 1]
			+ f[i + nx*j + nx*ny * 2] + f[i + nx*j + nx*ny * 3] + f[i + nx*j + nx*ny * 4]
			+ f[i + nx*j + nx*ny * 5] + f[i + nx*j + nx*ny * 6] + f[i + nx*j + nx*ny * 7]
			+ f[i + nx*j + nx*ny * 8];

		Ux[i + nx*j] = f[i + nx*j + nx*ny * 1] * ex[1] + f[i + nx*j + nx*ny * 3] * ex[3]
			+ f[i + nx*j + nx*ny * 5] * ex[5] + f[i + nx*j + nx*ny * 6] * ex[6] + f[i + nx*j + nx*ny * 7] * ex[7]
			+ f[i + nx*j + nx*ny * 8] * ex[8];

		Uy[i + nx*j] = f[i + nx*j + nx*ny * 2] * ey[2] + f[i + nx*j + nx*ny * 4] * ey[4]
			+ f[i + nx*j + nx*ny * 5] * ey[5] + f[i + nx*j + nx*ny * 6] * ey[6] + f[i + nx*j + nx*ny * 7] * ey[7]
			+ f[i + nx*j + nx*ny * 8] * ey[8];

		Ux[i + nx*j] /= rho[i + nx*j];
		Uy[i + nx*j] /= rho[i + nx*j];
		U[i + nx*j] = sqrt(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2));




		rhoN[i + nx*j] = fN[i + nx*j + nx*ny * 0] + fN[i + nx*j + nx*ny * 1]
			+ fN[i + nx*j + nx*ny * 2] + fN[i + nx*j + nx*ny * 3] + fN[i + nx*j + nx*ny * 4]
			+ fN[i + nx*j + nx*ny * 5] + fN[i + nx*j + nx*ny * 6] + fN[i + nx*j + nx*ny * 7]
			+ fN[i + nx*j + nx*ny * 8];

		UxN[i + nx*j] = fN[i + nx*j + nx*ny * 1] * ex[1] + fN[i + nx*j + nx*ny * 3] * ex[3]
			+ fN[i + nx*j + nx*ny * 5] * ex[5] + fN[i + nx*j + nx*ny * 6] * ex[6] + fN[i + nx*j + nx*ny * 7] * ex[7]
			+ fN[i + nx*j + nx*ny * 8] * ex[8];

		UyN[i + nx*j] = fN[i + nx*j + nx*ny * 2] * ey[2] + fN[i + nx*j + nx*ny * 4] * ey[4]
			+ fN[i + nx*j + nx*ny * 5] * ey[5] + fN[i + nx*j + nx*ny * 6] * ey[6] + fN[i + nx*j + nx*ny * 7] * ey[7]
			+ fN[i + nx*j + nx*ny * 8] * ey[8];

		UxN[i + nx*j] /= rhoN[i + nx*j];
		UyN[i + nx*j] /= rhoN[i + nx*j];
		UN[i + nx*j] = sqrt(pow(UxN[i + nx*j], 2) + pow(UyN[i + nx*j], 2));
	}
}
void LBM_GPU::Error() {

	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Error << < dimGrid, dimBlock >> > (d_f, d_Ux, d_Uy, d_U, d_rho, d_fN, d_UxN, d_UyN, d_UN, d_rhoN, d_ex, d_ey, nx, ny, a, d_is_solid_node);

	hipMemcpy(U, d_U, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(UN, d_UN, nx*ny * sizeof(float), hipMemcpyDeviceToHost);

	sum = 0.0;
	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {

			if (!is_solid_node[i + nx*j]) {
				sum = sum + pow(abs(UN[i + nx*j] - U[i + nx*j]), 2);
			}
		}
	}
	error = sqrt(sum / (nx*ny - sn));

}

__global__ 
void Kernel_Update(float* fN, float* f, float* Ux, float* Uy, float* U, float* rho, float* ex, float* ey, int nx, int ny, int a, int* is_solid_node) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.z * blockIdx.z + threadIdx.z;
	if (i >= nx || j >= ny || k >= a) return;

	if(!is_solid_node[i + nx*j]) f[i + nx*j + nx*ny*k] = fN[i + nx*j + nx*ny*k];

	rho[i + nx*j] = f[i + nx*j + nx*ny * 0] + f[i + nx*j + nx*ny * 1]
		+ f[i + nx*j + nx*ny * 2] + f[i + nx*j + nx*ny * 3] + f[i + nx*j + nx*ny * 4]
		+ f[i + nx*j + nx*ny * 5] + f[i + nx*j + nx*ny * 6] + f[i + nx*j + nx*ny * 7]
		+ f[i + nx*j + nx*ny * 8];

	Ux[i + nx*j] = f[i + nx*j + nx*ny * 1] * ex[1] + f[i + nx*j + nx*ny * 3] * ex[3]
		+ f[i + nx*j + nx*ny * 5] * ex[5] + f[i + nx*j + nx*ny * 6] * ex[6] + f[i + nx*j + nx*ny * 7] * ex[7]
		+ f[i + nx*j + nx*ny * 8] * ex[8];

	Uy[i + nx*j] = f[i + nx*j + nx*ny * 2] * ey[2] + f[i + nx*j + nx*ny * 4] * ey[4]
		+ f[i + nx*j + nx*ny * 5] * ey[5] + f[i + nx*j + nx*ny * 6] * ey[6] + f[i + nx*j + nx*ny * 7] * ey[7]
		+ f[i + nx*j + nx*ny * 8] * ey[8];

	Ux[i + nx*j] /= rho[i + nx*j];
	Uy[i + nx*j] /= rho[i + nx*j];
	U[i + nx*j] = sqrt(pow(Ux[i + nx*j], 2) + pow(Uy[i + nx*j], 2));
}
void LBM_GPU::Update() {


	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
	dim3 dimGrid((nx + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (ny + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y, (a + BLOCK_SIZE_Z - 1) / BLOCK_SIZE_Z);
	Kernel_Update << < dimGrid, dimBlock >> > (d_fN, d_f, d_Ux, d_Uy, d_U, d_rho, d_ex, d_ey, nx, ny, a, d_is_solid_node);

}

void LBM_GPU::Momentum() {
	hipMemcpy(f, d_f, nx*ny*a * sizeof(float), hipMemcpyDeviceToHost);

	sum_Fx1 = 0.0;
	sum_Fx3 = 0.0;
	sum_Fx5 = 0.0;
	sum_Fx6 = 0.0;
	sum_Fx7 = 0.0;
	sum_Fx8 = 0.0;
	sum_Fy2 = 0.0;
	sum_Fy4 = 0.0;
	sum_Fy5 = 0.0;
	sum_Fy6 = 0.0;
	sum_Fy7 = 0.0;
	sum_Fy8 = 0.0;

	sum_Fx = 0.0;
	sum_Fy = 0.0;

	Fx = 0.0;
	Fy = 0.0;

	Cd = 0.0;
	Cl = 0.0;

	for (i = 0; i < nx; i++) {
		for (j = 0; j < ny; j++) {

			sum_Fx1 = 0.0;
			sum_Fx3 = 0.0;
			sum_Fx5 = 0.0;
			sum_Fx6 = 0.0;
			sum_Fx7 = 0.0;
			sum_Fx8 = 0.0;
			sum_Fy2 = 0.0;
			sum_Fy4 = 0.0;
			sum_Fy5 = 0.0;
			sum_Fy6 = 0.0;
			sum_Fy7 = 0.0;
			sum_Fy8 = 0.0;

			sum_Fx = 0.0;
			sum_Fy = 0.0;

			in = i - 1;
			ip = i + 1;
			jn = j - 1;
			jp = j + 1;

			dist = sqrt(pow((float)i - ic, 2) + pow((float)j - jc, 2));
			q = dist - r;

			if (!is_boundary_node[i + nx*j]) {
				if (!is_solid_node[i + nx*j]) {
					if (is_solid_near_node[i + nx*j]) {


						if (is_solid_node[ip + nx*j]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 3] = 2.0 * q * f[i + nx*j + nx*ny * 1] + (1.0 - 2.0*q)*f[(i - 1) + nx*j + nx*ny * 1];
							else ftemp[i + nx*j + nx*ny * 3] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 1] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 3];

							sum_Fx1 = ex[1] * (ftemp[i + nx*j + nx*ny * 3] + f[i + nx*j + nx*ny * 1]);
						}

						if (is_solid_node[i + nx*jp]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 4] = 2.0 * q * f[i + nx*j + nx*ny * 2] + (1.0 - 2.0*q)*f[i + nx*(j - 1) + nx*ny * 2];
							else ftemp[i + nx*j + nx*ny * 4] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 2] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 4];

							sum_Fy2 = ey[2] * (ftemp[i + nx*j + nx*ny * 4] + f[i + nx*j + nx*ny * 2]);
						}

						if (is_solid_node[in + nx*j]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 1] = 2.0 * q * f[i + nx*j + nx*ny * 3] + (1.0 - 2.0*q)*f[(i + 1) + nx*j + nx*ny * 3];
							else ftemp[i + nx*j + nx*ny * 1] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 3] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 1];

							sum_Fx3 = ex[3] * (ftemp[i + nx*j + nx*ny * 1] + f[i + nx*j + nx*ny * 3]);
						}

						if (is_solid_node[i + nx*jn]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 2] = 2.0 * q * f[i + nx*j + nx*ny * 4] + (1.0 - 2.0*q)*f[i + nx*(j + 1) + nx*ny * 4];
							else ftemp[i + nx*j + nx*ny * 2] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 4] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 2];

							sum_Fy4 = ey[4] * (ftemp[i + nx*j + nx*ny * 2] + f[i + nx*j + nx*ny * 4]);
						}

						if (is_solid_node[ip + nx*jp]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 7] = 2.0 * q * f[i + nx*j + nx*ny * 5] + (1.0 - 2.0*q)*f[(i - 1) + nx*(j - 1) + nx*ny * 5];
							else ftemp[i + nx*j + nx*ny * 7] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 5] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 7];

							sum_Fx5 = ex[5] * (ftemp[i + nx*j + nx*ny * 7] + f[i + nx*j + nx*ny * 5]);
							sum_Fy5 = ey[5] * (ftemp[i + nx*j + nx*ny * 7] + f[i + nx*j + nx*ny * 5]);
						}

						if (is_solid_node[in + nx*jp]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 8] = 2.0 * q * f[i + nx*j + nx*ny * 6] + (1.0 - 2.0*q)*f[(i + 1) + nx*(j - 1) + nx*ny * 6];
							else ftemp[i + nx*j + nx*ny * 8] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 6] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 8];

							sum_Fx6 = ex[6] * (ftemp[i + nx*j + nx*ny * 8] + f[i + nx*j + nx*ny * 6]);
							sum_Fy6 = ey[6] * (ftemp[i + nx*j + nx*ny * 8] + f[i + nx*j + nx*ny * 6]);
						}

						if (is_solid_node[in + nx*jn]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 5] = 2.0 * q * f[i + nx*j + nx*ny * 7] + (1.0 - 2.0*q)*f[(i + 1) + nx*(j + 1) + nx*ny * 7];
							else ftemp[i + nx*j + nx*ny * 5] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 7] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 5];

							sum_Fx7 = ex[7] * (ftemp[i + nx*j + nx*ny * 5] + f[i + nx*j + nx*ny * 7]);
							sum_Fy7 = ey[7] * (ftemp[i + nx*j + nx*ny * 5] + f[i + nx*j + nx*ny * 7]);
						}

						if (is_solid_node[ip + nx*jn]) {

							if (q < 0.5) ftemp[i + nx*j + nx*ny * 6] = 2.0 * q * f[i + nx*j + nx*ny * 8] + (1.0 - 2.0*q)*f[(i - 1) + nx*(j + 1) + nx*ny * 8];
							else ftemp[i + nx*j + nx*ny * 6] = (1.0 / (2.0*q))*f[i + nx*j + nx*ny * 8] + (2.0*q - 1.0) / (2.0*q)*f[i + nx*j + nx*ny * 6];

							sum_Fx8 = ex[8] * (ftemp[i + nx*j + nx*ny * 6] + f[i + nx*j + nx*ny * 8]);
							sum_Fy8 = ey[8] * (ftemp[i + nx*j + nx*ny * 6] + f[i + nx*j + nx*ny * 8]);
						}


						sum_Fx = sum_Fx1 + sum_Fx3 + sum_Fx5 + sum_Fx6 + sum_Fx7 + sum_Fx8;
						sum_Fy = sum_Fy2 + sum_Fy4 + sum_Fy5 + sum_Fy6 + sum_Fy7 + sum_Fy8;


						Fx = Fx + sum_Fx;
						Fy = Fy + sum_Fy;


					}

				}
			}
		}
	}

	Cd = 2.0*Fx / (rho1*pow((2.0 / 3.0)*Um, 2)*snx);
	Cl = 2.0*Fy / (rho1*pow((2.0 / 3.0)*Um, 2)*sny);

	fout_GPU_Cd << Cd << "\t" << Cl << endl;
}

void LBM_GPU::Print() {

	hipMemcpy(Ux, d_Ux, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Uy, d_Uy, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(U, d_U, nx*ny * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(rho, d_rho, nx*ny * sizeof(float), hipMemcpyDeviceToHost);


// ============================================================================ //
//  CHANGE LBM -> PHYSICAL
// ============================================================================ //
	for (j = 0; j < ny; j++) {
		for (i = 0; i < nx; i++) {
			Ux_p[i + nx*j] = Ux[i + nx*j];
			Uy_p[i + nx*j] = Uy[i + nx*j];
			U_p[i + nx*j] = U[i + nx*j];
			P[i + nx*j] = rho[i + nx*j] / (3.0);
		}
	}
// ============================================================================ //



	fout_GPU << endl;
	fout_GPU << "variables = X Y Ux Uy U rho P" << endl;
	fout_GPU << "zone i=" << nx << " j=" << ny << endl;
	for (j = 0; j < ny; j++) {
		for (i = 0; i < nx; i++) {
			fout_GPU << i << "\t" << j << "\t" << Ux[i + nx*j] << "\t" << Uy[i + nx*j] << "\t"
				<< U[i + nx*j] << "\t" << rho[i + nx*j] << "\t" << P[i + nx*j] << endl;
		}
	}
	fout_GPU << endl;


	i = 0;
	fout_GPU_Ux << "variables = X Y Ux" << endl;
	fout_GPU_Ux << "zone i=" << nx << " j=" << ny << endl;
	for (j = 0; j < ny; j++) {

		fout_GPU_Ux << i << "\t" << j << "\t" << Ux[i + nx*j] << endl;

		
	}
	fout_GPU_Ux << endl;
}

LBM_GPU::~LBM_GPU()
{
	hipFree(d_Ux0);
	hipFree(d_is_boundary_node);
	hipFree(d_is_solid_node);
	hipFree(d_f);
	hipFree(d_fN);
	hipFree(d_ftemp);
	hipFree(d_feq);
	hipFree(d_Ux);
	hipFree(d_Uy);
	hipFree(d_rho);
	hipFree(d_ex);
	hipFree(d_ey);
	hipFree(d_U);
	hipFree(d_UN);
	hipFree(d_UxN);
	hipFree(d_UyN);
	hipFree(rhoN);

	delete[] Ux0;
	delete[] Ux0_p;
	delete[] P;
	delete[] Uy_p;
	delete[] Ux_p;
	delete[] U_p;
	delete[] ey;
	delete[] ex;
	delete[] fN;
	delete[] feq;
	delete[] ftemp;
	delete[] f;
	delete[] rhoN;
	delete[] UyN;
	delete[] UxN;
	delete[] UN;
	delete[] rho;
	delete[] Uy;
	delete[] Ux;
	delete[] U;
	delete[] is_boundary_node;
	delete[] is_solid_node;
	delete[] is_solid_near_node;
	cout << endl << "Done!" << endl;
}
